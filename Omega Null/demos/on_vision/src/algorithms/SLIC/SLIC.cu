#include "hip/hip_runtime.h"

#include"vision_manifold.h"
#include"../../vision.h"
#include"SLIC.h"


#pragma region sample_centers

	__global__ void sample_centers(on::Tensor<int> source, on::Tensor<int> center_pos) {
		GET_DIMS(maj, min);
		CHECK_BOUNDS(center_pos.maj_span, center_pos.min_span);

		center_pos(maj, min, 0) = CAST_UP(maj, center_pos.maj_span, source.maj_span);
		center_pos(maj, min, 1) = CAST_UP(min, center_pos.min_span, source.min_span);

	}

	__global__ void gradient_descent(on::Tensor<int> source, on::Tensor<int> center_pos) {
		GET_DIMS(maj, min);
		CHECK_BOUNDS(center_pos.maj_span, center_pos.min_span);

		//gradient descent

	}

#pragma endregion

#pragma region assign_pixels_to_centers

	__global__ void pixels_to_centers(on::Tensor<int> source, on::Tensor<int> center_pos, int distance_modifier, on::Tensor<int> flags) {
		GET_DIMS(maj, min);
		CHECK_BOUNDS(source.maj_span, source.min_span);

		int sector_maj = CAST_DOWN(maj, center_pos.maj_span);
		int sector_min = CAST_DOWN(min, center_pos.min_span);

		int self_channels[3] = {source(maj, min, 0), source(maj, min, 1), source(maj, min, 2)};
		int self_position[2] = {maj, min};

		int closest_center = -1;
		int smallest_distance = INT_MAX;

		FOR_3X3_INCLUSIVE(n_maj, n_min, center_pos.maj_span, center_pos.min_span, maj, min, 
			int center_id = LINEAR_CAST(n_maj, n_min, center_pos.min_span);
			int neighbor_maj = center_pos(n_maj, n_min, 0);
			int neighbor_min = center_pos(n_maj, n_min, 1);

			int neighbor_channels[3] = {source(neighbor_maj, neighbor_min, 0), source(neighbor_maj, neighbor_min, 1), source(neighbor_maj, neighbor_min, 2)};
			int neighbor_position[2] = {neighbor_maj, neighbor_min};

			int color_distance = 0;
			for (int channel = 0; channel < 3; channel++) {
				color_distance += (neighbor_channels[channel] - self_channels[channel]) * (neighbor_channels[channel] - self_channels[channel]);
			}
		
			int spatial_distance = 0;
			for (int coordinate = 0; coordinate < 2; coordinate++) {
				spatial_distance += (neighbor_position[coordinate] - neighbor_position[coordinate]) * (neighbor_position[coordinate] - neighbor_position[coordinate]);
			}

			int neighbor_distance = color_distance + (distance_modifier * spatial_distance);

			if (neighbor_distance < smallest_distance) {
				closest_center = center_id;
				smallest_distance = neighbor_distance;
			}
		);

		int result = closest_center;
		flags(maj, min) = result; 
		return;
	}

#pragma endregion

#pragma region update_centers
	
	__global__ void tally_centers(on::Tensor<int> flags, on::Tensor<int> tally) {
		GET_DIMS(maj, min);
		CHECK_BOUNDS(flags.maj_span, flags.min_span);

		int id = flags(maj, min);

		atomicAdd(&tally(id, 0), maj);
		atomicAdd(&tally(id, 1), min);
		atomicAdd(&tally(id, 2), 1);

	}

	__global__ void move_centers(on::Tensor<int> tally, on::Tensor<int> center_pos, int* displacement) {
		GET_DIMS(id, ZERO);
		CHECK_BOUNDS(tally.maj_span, 1);
		
		int maj = id % center_pos.min_span;
		int min = (id - min) / center_pos.min_span;

		int old_maj = center_pos(maj, min, 0);
		int old_min = center_pos(maj, min, 1);

		int maj_sum = tally(id, 0);
		int min_sum = tally(id, 1);
		int size = tally(id, 2);

		int new_maj = maj_sum / size;
		int new_min = min_sum / size;

		int this_displacement = sqrtf(((old_maj - new_maj) * (old_maj - new_maj)) + ((old_min - new_min) * (old_min - new_min)));

		atomicAdd(displacement, this_displacement);

		center_pos(min, maj, 0) = new_maj;
		center_pos(min, maj, 1) = new_min;
		
	}

#pragma endregion

#pragma region separate_blobs


#pragma endregion

#pragma region absorb_small_blobs


#pragma endregion

#pragma region produce_ordered_labels


#pragma endregion

namespace on {

	On_Structure Vision{

		On_Structure Algorithm {
			
			using namespace Parameter::SLIC;
			On_Process SLIC {
				

				void SLIC::sample_centers(Tensor<int>& source, Tensor<int>& center_pos) {

					Launch::Kernel::conf_2d(center_pos.maj_span, center_pos.min_span);
					sample_centers<<<LAUNCH>>>(source, center_pos);
					On_Sync(sample_centers);

					Launch::Kernel::conf_2d(center_pos.maj_span, center_pos.min_span);
					gradient_descent<<<LAUNCH>>>(source, center_pos);
					On_Sync(gradient_descent);

				}

				void SLIC::assign_pixels_to_centers(Tensor<int>& source, Tensor<int>& center_pos, Tensor<int>& flags) {

					Launch::Kernel::conf_2d(source.maj_span, source.min_span);
					pixels_to_centers<<<LAUNCH>>>(source, center_pos, , flags);
					On_Sync(pixels_to_centers);

				}

				void SLIC::update_centers(Tensor<int>& flags, Tensor<int>& center_pos) {

					//0 = maj sum , 1 = min sum, 2 = count
					on::Tensor<int> tally({(uint)Parameter::SLIC::num_superpixels, 3});

					Launch::Kernel::conf_2d(flags.maj_span, flags.min_span);
					tally_centers<<<LAUNCH>>>(flags, tally);
					On_Sync(tally_centers);

					int* temp_d_displacement;
					hipMalloc(&temp_d_displacement, sizeof(int));

					set_flag(temp_displacement);

					Launch::Kernel::conf_1d(tally.maj_span);
					move_centers<<<LAUNCH>>>(tally, center_pos, temp_d_displacement); 
					On_Sync(update_centers);

					get_flag(temp_displacement);

					int temp_h_displacement;
					hipMemcpy(&temp_h_displacement, temp_d_displacement, sizeof(int), hipMemcpyDeviceToHost);
					hipFree(temp_d_displacement);

					Parameter::SLIC::displacement = temp_h_displacement;
					
				}

				#pragma region enforce connectivity
					void SLIC::separate_blobs() {

					}

					void SLIC::absorb_small_blobs() {

					}

					void SLIC::produce_ordered_labels() {

					}
				#pragma endregion

				void SLIC::enforce_connectivity(on::Tensor<int>& flags) {
					separate_blobs();
					absorb_small_blobs();
					produce_ordered_labels();
				}

				void SLIC::run(Clip<int>& input, Clip<int>& output) {
					for (Tensor source : input.frames) {

						source_maj = source.maj_span;
						source_min = source.min_span;
						num_pixels = source_maj * source_min;

						SP_maj;
						SP_min;
						num_superpixels;

						space_between_centers;
						density_modifier;

						Tensor<int> flags({ (uint)source_maj, (uint)source_min }, 0);

						//z = 0 is maj, z = 1 is min
						Tensor<int> center_pos({(uint)SP_maj, (uint)SP_min, (uint)2}, 0);


						sample_centers(center_pos, source);

						do {

							assign_pixels_to_centers(source, center_pos, flags);

							update_centers(flags, center_pos);

						} while (Parameter::SLIC::displacement < Parameter::SLIC::displacement_threshold);

						enforce_connectivity(flags);

						output.frames.push_back(flags);
					}
					return;
				}
			};
		}
	}

}