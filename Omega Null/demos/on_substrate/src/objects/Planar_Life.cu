#include "hip/hip_runtime.h"

#include"substrate_manifold.h"
#include"omega_null.h"
#include"Planar_Life.h"




namespace Cell {
	enum attribute {
		color_r = 0,
		color_g = 1,
		color_b = 2,
		attractor = 3,
		weight = 4,
		freq_a = 5,
		value = 6,
		move_maj = 7,
		move_min = 8,
	};

	const int num_attributes = 9;

}; using namespace Cell;


__global__ void draw_environment(sk::Device_Ptr<int> environment, sk::Device_Ptr<uchar> output) {
	DIMS_2D(maj, min);
	BOUNDS_2D(environment.maj(), environment.min());

	int environment_value = environment(maj, min);

	int polarity[3] = {0,0,0};

	if (environment_value > 0) { //positive pressure becomes red tinted
		polarity[0] += 20;
	}

	else if (environment_value < 0) { //negative pressure becomes blue tinted
		polarity[2] += 20;
	}

	int magnitude = fabsf(environment_value);

	for (int channel = 0; channel < 3; channel++) {
		output.device_data[(((channel * environment.maj()) + maj) * environment.min()) + min] = fminf(logf(10 * magnitude), 90) + polarity[channel];
	}
}


//pretty goofy way to do this to be honest. But let's see how fast or slow it runs.
__global__ void draw_cells(sk::Device_Ptr<int> cells, sk::Device_Ptr<uchar> output) {
	DIMS_2D(maj, min);
	BOUNDS_2D(cells.maj(), cells.min());

	if(cells(SELF, attribute::color_r) == 0 && cells(SELF, attribute::color_g) == 0 && cells(SELF, attribute::color_b) == 0) { return; }

	for (int channel = 0; channel < 3; channel++) {
		int channel_value = cells(maj, min, channel);

		output.device_data[(((channel * cells.maj()) + maj ) * cells.min()) + min] = channel_value;
	}
}



#define Random(_seed_, _min_, _max_) (((_seed_) % ((_max_) - (_min_)))+ (_min_))

__global__ void spawn(sk::Device_Ptr<int> mask, hiprandState* random_states, sk::Device_Ptr<int> result) {
	DIMS_2D(maj, min);
	BOUNDS_2D(result.maj(), result.min());
	//if (mask(maj, min) == 0) { return; } //mask is broken somehow, this seems to delete ALL threads rather than just half.

	int id = LINEAR_CAST(maj, min, result.min());

	hiprandState local_state = random_states[id];

	int random_0 = hiprand(&local_state);
	int random_1 = hiprand(&local_state);
	//int random_2 = hiprand(&local_state);
	int random_3 = hiprand(&local_state);
	int random_4 = hiprand(&local_state);
	int random_5 = hiprand(&local_state); 
	int random_6 = hiprand(&local_state);

	int attractor = Random(random_0, -10000, 10000);
	int weight = Random(random_1, -1000, 1000);
	int value = 50;
	int color_r = Random(random_3, 70, 255);
	int color_g = Random(random_4, 70, 255);
	int color_b = Random(random_5, 70, 255);
	int freq_a = Random(random_6, -10000, 10000);

	result(SELF, attribute::value) = value;
	result(SELF, attribute::attractor) = attractor;
	result(SELF, attribute::weight) = weight;
	result(SELF, attribute::move_maj) = maj;
	result(SELF, attribute::move_min) = min;
	result(SELF, attribute::color_r) = color_r;
	result(SELF, attribute::color_g) = color_g;
	result(SELF, attribute::color_b) = color_b;
	result(SELF, attribute::freq_a) = freq_a;
}

__global__ void change_environment(sk::Device_Ptr<int> environment, sk::Device_Ptr<int> cells) {
	DIMS_2D(maj, min);
	BOUNDS_2D(environment.maj(), environment.min());

	int weight = (cells(SELF, attribute::value) * cells(SELF, attribute::weight))/10;
	FOR_MXN_INCLUSIVE(n_maj, n_min, 9, 9, environment.maj(), environment.min(), maj, min,
		atomicAdd(&environment(n_maj, n_min), weight);
	);
}

__global__ void dampen_environment(const float damping_factor, sk::Device_Ptr<int> environment) {
	DIMS_2D(maj, min);
	BOUNDS_2D(environment.maj(), environment.min());

	float value = environment(maj, min);
	environment(maj, min) = truncf(value * damping_factor);


}

__global__ void radiate_environment(sk::Device_Ptr<int> environment) {
	DIMS_2D(maj, min);
	BOUNDS_2D(environment.maj(), environment.min());

	int value = environment(SELF);
	FOR_NEIGHBOR(n_maj, n_min, environment.maj(), environment.min(), maj, min, 
		
		atomicSub(&environment(n_maj, n_min), value/10);
	
	)
}

__global__ void set_targets(sk::Device_Ptr<int> environment, sk::Device_Ptr<int> cells, sk::Device_Ptr<int> targets) {
	DIMS_2D(maj, min);
	BOUNDS_2D(environment.maj(), environment.min());


	int attractor = cells(SELF, attribute::attractor);
	int weight = cells(SELF, attribute::weight);
	if(attractor == 0 && weight == 0){return;}

	int largest_value = environment(maj, min) * attractor;
	//int largest_value = -1;
	int target_maj = maj;
	int target_min = min;

	FOR_NEIGHBOR(n_maj, n_min, environment.maj(), environment.min(), maj, min,
		int neighbor_value = environment(n_maj, n_min) * attractor;
		if ( neighbor_value > largest_value) {
			largest_value = neighbor_value;
			target_maj = n_maj;
			target_min = n_min;
		}
	);

	atomicAdd(&targets(target_maj, target_min), 1);
	cells(SELF, attribute::move_maj) = target_maj;
	cells(SELF, attribute::move_min) = target_min;
}

__global__ void conflict(sk::Device_Ptr<int>cells, sk::Device_Ptr<int>targets, sk::Device_Ptr<int>future_cells, const int threshold = 3) {
	
	DIMS_2D(maj, min);
	BOUNDS_2D(cells.maj(), cells.min());

	if(targets(maj, min) < 2) {return;}

	int teams[9][Cell::num_attributes - 2] = {0};
	int num_teams = 0;

	FOR_3X3_INCLUSIVE(n_maj, n_min, cells.maj(), cells.min(), maj, min, 
		
		if((cells(n_maj, n_min, attribute::move_maj) != maj)|| (cells(n_maj, n_min, attribute::move_min) != min)) {continue;}

		bool unassigned = true;
		for (int team = 0; team < num_teams; team++) {
			int difference = fabsf(teams[team][attribute::freq_a] - cells(n_maj, n_min, attribute::freq_a));
			if (difference < threshold) { continue; }

			unassigned = false;

			teams[team][attribute::value] += cells(n_maj, n_min, attribute::value);

			for (int attribute = 2; attribute < Cell::num_attributes - 3; attribute++) {
				teams[team][attribute] += cells(n_maj, n_min, attribute);
				teams[team][attribute] /= 2;
			}

		}
		
		for (int attribute = 0; attribute < Cell::num_attributes - 2; attribute++) {
			teams[num_teams][attribute] += cells(n_maj, n_min, attribute) * unassigned;
		}
		num_teams += unassigned
	);

	int highest_value = -1;
	int winning_team = -1;
	int total_value = 0;

	for (int i = 0; i < 9; i++) {
		total_value += teams[i][attribute::value];
		if (teams[i][attribute::value] > highest_value) {
			highest_value = teams[i][attribute::value];
			winning_team = i;
		}
	}

	future_cells(SELF, attribute::value) = total_value;
	for (int i = 0; i < Cell::num_attributes - 3; i++) {
		future_cells(SELF, i) = teams[winning_team][i];
	}
}

__global__ void move(sk::Device_Ptr<int> environment, sk::Device_Ptr<int> cells, sk::Device_Ptr<int> targets, sk::Device_Ptr<int> future_cells) {
	DIMS_2D(maj, min);
	BOUNDS_2D(environment.maj(), environment.min());

	int attractor = cells(SELF, attribute::attractor);
	int weight = cells(SELF, attribute::weight);
	int target_maj = cells(SELF, attribute::move_maj);
	int target_min = cells(SELF, attribute::move_min);

	if(targets(target_maj, target_min) != 1){return;}
	if (attractor == 0 && weight == 0) { return; }

	//#pragma unroll
	for (int i = 0; i < Cell::num_attributes; i++) {
		future_cells(target_maj, target_min, i) = cells(SELF, i);
	}

}

__global__ void hatch(const int threshold, hiprandState* random_states, sk::Device_Ptr<int> cells) {
	DIMS_2D(maj, min);
	BOUNDS_2D(cells.maj(), cells.min());

	int value = cells(SELF, attribute::value);
	if (value < threshold){return;} 

	int random[Cell::num_attributes];
	int id = LINEAR_CAST(maj, min, cells.min());
	hiprandState local_state = random_states[id];

	for (int i = 0; i < Cell::num_attributes; i++) {
		random[i] = hiprand(&local_state);
	}

	FOR_NEIGHBOR(n_maj, n_min, cells.maj(), cells.min(), maj, min, 
		if(cells(n_maj, n_min, attribute::attractor) == 0 && cells(n_maj, n_min, attribute::weight) == 0){
			cells(SELF, attribute::value) -= 5;

			cells(n_maj, n_min, attribute::color_r) = fmaxf( 70, fminf(cells(SELF, attribute::color_r), 255));
			cells(n_maj, n_min, attribute::color_g) = fmaxf(70, fminf(cells(SELF, attribute::color_g), 255));
			cells(n_maj, n_min, attribute::color_b) = fmaxf(70, fminf(cells(SELF, attribute::color_b), 255));
			cells(n_maj, n_min, attribute::attractor) = fmaxf(-10000, fminf(cells(SELF, attribute::attractor) + Random(random[attribute::attractor], -1, 1), 10000 ));
			cells(n_maj, n_min, attribute::weight) = fmaxf(-1000, fminf(cells(SELF, attribute::weight) + Random(random[attribute::weight], -1, 1), 1000));
			cells(n_maj, n_min, attribute::freq_a) = fmaxf(-10000, fminf(cells(SELF, attribute::freq_a) + Random(random[attribute::freq_a], -1, 1), 10000) );
			cells(n_maj, n_min, attribute::value) = 5;
			return;
		}
	);

	//cells(SELF, attribute::value) /= 2;
}

namespace on {

	On_Structure Substrate {

		On_Structure Species {

			On_Structure Planar_Life {

				sk::Tensor<int> Seed::cells(int value = 0) {

					sk::Tensor<int> result({Parameter::environment_width, Parameter::environment_height, Cell::num_attributes}, 0, "result");
					af::array af_mask = (af::randu(Parameter::environment_width, Parameter::environment_height) > 0.5).as(s32);
					sk::Tensor<int> mask({Parameter::environment_width, Parameter::environment_height}, 0, "mask");
					//mask = af_mask; //possible failure here

					hiprandState* states = on::Random::Initialize::curand_xor(Parameter::environment_area, value);

					sk::configure::kernel_2d(result.maj(), result.min());
					spawn<<<LAUNCH>>>(mask, states, result); //try using the nvidia debugger
					SYNC_KERNEL(spawn); 

					hipFree(states); //bad way of doing this, because it's not clear that one would have to call cudafree on curand_xor. should at least put it in on::Random::Delete

					return result;

				}

				sk::Tensor<uchar> Draw::frame(sk::Tensor<int>& cells, sk::Tensor<int>& environment) {

					sk::Tensor<uchar> output({cells.maj(), cells.min(), 3}, 0);

					sk::configure::kernel_2d(cells.maj(), cells.min());
					//draw_environment<<<LAUNCH>>>(environment, output);
					//SYNC_KERNEL(draw_environment);

					draw_cells <<<LAUNCH>>> (cells, output);
					SYNC_KERNEL(draw_cells);

					return output;

				}

				void Planar_Life::Step::polar(sk::Tensor<int>& future_cells, sk::Tensor<int>& environment, sk::Tensor<int>& cells, sk::Tensor<int>& targets, hiprandState* random) {
					sk::configure::kernel_2d(environment.maj(), environment.min());
					
					const int thresh = 6;
					hatch << <LAUNCH >> > (thresh, random, cells);
					SYNC_KERNEL(hatch);

					change_environment<<<LAUNCH>>> (environment, cells); 
					SYNC_KERNEL(change_environment);

					radiate_environment << <LAUNCH >> > (environment);
					SYNC_KERNEL(radiate_environment);

					const float damping_factor = 0.3;
					dampen_environment<<<LAUNCH>>>(damping_factor, environment);
					SYNC_KERNEL(dampen_environment);



					set_targets<<<LAUNCH>>>(environment, cells, targets);
					SYNC_KERNEL(set_targets);

					const int threshold = 500;
					conflict<<<LAUNCH>>>(cells, targets, future_cells, threshold);
					SYNC_KERNEL(conflict);

					move<<<LAUNCH>>> (environment, cells, targets, future_cells); 
					SYNC_KERNEL(move);



					cells = future_cells;
				}
			}
		}
	}
}












